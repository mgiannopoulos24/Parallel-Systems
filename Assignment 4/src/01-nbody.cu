#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include "timer.h"
#include "files.h"
#include <hip/hip_runtime.h>

#define SOFTENING 1e-9f

/*
 * Each body contains x, y, and z coordinate positions,
 * as well as velocities in the x, y, and z directions.
 */

typedef struct { float x, y, z, vx, vy, vz; } Body;

/*
 * CUDA kernel to calculate the gravitational impact of all bodies in the system
 * on all others.
 */

__global__ void bodyForceKernel(Body *p, float dt, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        float Fx = 0.0f; float Fy = 0.0f; float Fz = 0.0f;
        float xi = p[i].x, yi = p[i].y, zi = p[i].z;

        for (int j = 0; j < n; j++) {
            float dx = p[j].x - xi;
            float dy = p[j].y - yi;
            float dz = p[j].z - zi;
            float distSqr = dx*dx + dy*dy + dz*dz + SOFTENING;
            float invDist = rsqrtf(distSqr);
            float invDist3 = invDist * invDist * invDist;

            Fx += dx * invDist3; Fy += dy * invDist3; Fz += dz * invDist3;
        }

        p[i].vx += dt*Fx; p[i].vy += dt*Fy; p[i].vz += dt*Fz;
    }
}

/*
 * CUDA kernel to integrate positions
 */

__global__ void integratePositionKernel(Body *p, float dt, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        p[i].x += p[i].vx*dt;
        p[i].y += p[i].vy*dt;
        p[i].z += p[i].vz*dt;
    }
}

int main(const int argc, const char** argv) {

    // The assessment will test against both 2<11 and 2<15.
    // Feel free to pass the command line argument 15 when you generate ./nbody report files
    int nBodies = 2<<11;
    if (argc > 1) nBodies = 2<<atoi(argv[1]);

    // The assessment will pass hidden initialized values to check for correctness.
    // You should not make changes to these files, or else the assessment will not work.
    const char * initialized_values;
    const char * solution_values;

    if (nBodies == 2<<11) {
        initialized_values = "09-nbody/files/initialized_4096";
        solution_values = "09-nbody/files/solution_4096";
    } else { // nBodies == 2<<15
        initialized_values = "09-nbody/files/initialized_65536";
        solution_values = "09-nbody/files/solution_65536";
    }

    if (argc > 2) initialized_values = argv[2];
    if (argc > 3) solution_values = argv[3];

    const float dt = 0.01f; // Time step
    const int nIters = 10;  // Simulation iterations

    int bytes = nBodies * sizeof(Body);
    float *buf;

    buf = (float *)malloc(bytes);

    Body *p = (Body*)buf;

    read_values_from_file(initialized_values, buf, bytes);

    // Allocate device memory
    Body *d_p;
    hipMalloc((void**)&d_p, bytes);
    hipMemcpy(d_p, p, bytes, hipMemcpyHostToDevice);

    double totalTime = 0.0;

    // Define block size and grid size
    int blockSize = 256;
    int gridSize = (nBodies + blockSize - 1) / blockSize;

    for (int iter = 0; iter < nIters; iter++) {
        StartTimer();

        // Launch bodyForce kernel
        bodyForceKernel<<<gridSize, blockSize>>>(d_p, dt, nBodies);

        // Launch integratePosition kernel
        integratePositionKernel<<<gridSize, blockSize>>>(d_p, dt, nBodies);

        hipDeviceSynchronize(); // Ensure kernels complete before timing

        const double tElapsed = GetTimer() / 1000.0;
        totalTime += tElapsed;
    }

    // Copy results back to host
    hipMemcpy(p, d_p, bytes, hipMemcpyDeviceToHost);

    double avgTime = totalTime / (double)(nIters);
    float billionsOfOpsPerSecond = 1e-9 * nBodies * nBodies / avgTime;
    write_values_to_file(solution_values, buf, bytes);

    printf("%0.3f Billion Interactions / second\n", billionsOfOpsPerSecond);

    // Free device memory
    hipFree(d_p);
    free(buf);

    return 0;
}